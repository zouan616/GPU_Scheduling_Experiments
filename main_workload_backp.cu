#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include ""
#include <stdlib.h>      
#include <time.h>  
#include <string.h>
#include <limits.h>
#include <math.h>
#include <unistd.h>
#include <fcntl.h>
#include <float.h>
#include <sys/time.h>
#include <fstream>
#include "timespec_functions.h"
#include <algorithm>
#include <pthread.h>
#include <errno.h>
#include <vector>
#include <assert.h>


using namespace std;
void * pthread0(void *data);
void * scheduler(void *data);

struct para
{
int task_num;
int iter;
int memory_length;
int kernel_length;
dim3 gridsize;
dim3 blocksize;
int SM_num_start;
int SM_num_end;
float *d_data01;
float *d_data02;
float *d_data03;
float *d_data1;
float *d_data2;
float *d_data3;
float *d_result;
long int N;
long int nBytes;
};

struct task
{
int task_num;
bool ready;
bool memory_finish;
bool kernel_finish;
};




//task number
int n = 8; 
struct task GPU_task[8];

struct timeval global_tv[8];
double global_start_time[8];
double global_memory_start_time[8];
double global_memory_finish_time[8];
double global_kernel_start_time[8];
double global_kernel_finish_time[8];
double global_duration[8];

// Scheduler order


int sched_order[] = {1};






struct timeval offset_tv;
double offset_start_time;


#include "task.cu"


inline hipError_t checkCuda(hipError_t result)
{
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}


//main function
int main(int argc,char *argv[])
{
    cpu_set_t cpuset0;
    CPU_ZERO(&cpuset0);
    CPU_SET(0, &cpuset0);
    int s;
    s = pthread_setaffinity_np(pthread_self(), sizeof(cpu_set_t), &cpuset0);
    if (s != 0)
        cout << "Fail to pin to core 0" << endl;

    //Initialize task queque


    int memory_length[n];
    int kernel_length[n];
    int CTA_num[n];

    for(int i = 0; i < n; i++)
    {
    GPU_task[i].task_num = i;
    GPU_task[i].ready = false;
    GPU_task[i].memory_finish = false;
    GPU_task[i].kernel_finish = false;
    }


    //create CPU thread
    pthread_t tidp[n];
    pthread_t scheduler_thread;



    //data length
    long int N = 1 << 18;



    ifstream config_input_parameter(argv[1],ios::app);
    

    //cout << argv[1] << endl;

    for(int i = 0; i < n; i++)
    {
    config_input_parameter >> memory_length[i];
    config_input_parameter >> kernel_length[i];
    config_input_parameter >> CTA_num[i];
    }
    



    config_input_parameter.close();

    int CTA_num_start[n], CTA_num_end[n];



     long int nBytes = N * sizeof(float);


    //Apply for host memory
    
    float *x[n], *y[n], *z[n];

    for(int i = 0; i < n; i++)
    {
    checkCuda(hipHostMalloc((void **) &x[i], nBytes));
    checkCuda(hipHostMalloc((void **) &y[i], nBytes));
    checkCuda(hipHostMalloc((void **) &z[i], nBytes));
        //Initialize data
        for(int j = 0; j < N; j++)
        {
          x[i][j] = j % 20;
          y[i][j] = j % 20;
          z[i][j] = 0;
        }
        
    }



    //Apply for GPU memory
    float *d_x[n], *d_y[n], *d_z[n];

    for(int i = 0; i < n; i++)
    {
    checkCuda(hipMalloc((void **) &d_x[i], nBytes));
    checkCuda(hipMalloc((void **) &d_y[i], nBytes));
    checkCuda(hipMalloc((void **) &d_z[i], nBytes));       
    }

    //Initial grid size

    dim3 blocksize(1024);
    dim3 gridsize(56);   

    //passing data
    struct para GPU_para[n];
    for(int i = 0; i < n; i++)
    {
    GPU_para[i].task_num = i;
    GPU_para[i].gridsize = gridsize;
    GPU_para[i].blocksize = blocksize;
    GPU_para[i].memory_length = memory_length[i];
    GPU_para[i].kernel_length = kernel_length[i];
    GPU_para[i].SM_num_start = CTA_num_start[i];
    GPU_para[i].SM_num_end = CTA_num_end[i];
    GPU_para[i].d_data01 = x[i];
    GPU_para[i].d_data02 = y[i];
    GPU_para[i].d_data03 = z[i];
    GPU_para[i].d_data1 = d_x[i];
    GPU_para[i].d_data2 = d_y[i];
    GPU_para[i].d_data3 = d_z[i];
    GPU_para[i].N = N;
    GPU_para[i].nBytes = nBytes;
    }

    

 

      

    for(int i = 0; i < n; i++)
    {
      pthread_create(&tidp[i], NULL, pthread0, (void *)& GPU_para[i]);
    }


    usleep(1000000);

    pthread_create(&scheduler_thread, NULL, scheduler, NULL);

    
    gettimeofday(&offset_tv,NULL);
    offset_start_time = offset_tv.tv_sec*1000 + offset_tv.tv_usec/1000;
    
    
    for(int i = 0; i < n; i++)
    {
    pthread_join(tidp[i],NULL);    
    }


    for(int i = 0; i < n; i++)
    {
    //cout << "-----------------------------" << endl;
    //cout << "task " << i+1 << " memory start time: " << global_memory_start_time[i] << endl;
    //cout << "task " << i+1 << " memory finish time: " << global_memory_finish_time[i] << endl;
    //cout << "task " << i+1 << " kernel start time: " << global_kernel_start_time[i] << endl;
    //cout << "task " << i+1 << " kernel finish time: " << global_kernel_finish_time[i] << endl;

    cout << global_kernel_finish_time[i] - global_kernel_start_time[i] << endl;
    }


    //Free device memory
    for(int i = 0; i < n; i++)
    {
    hipFree(d_x[i]);
    hipFree(d_y[i]);
    hipFree(d_z[i]);      
    }



    //Free host memory
    for(int i = 0; i < n; i++)
    {
    hipHostFree(x[i]);
    hipHostFree(y[i]);
    hipHostFree(z[i]); 
    }

    //cout << "Finish!" << endl;
    return 0;
}



void * scheduler(void *data)       
{

    

    // pin to a core
    cpu_set_t cpuset1;
    CPU_ZERO(&cpuset1);
    CPU_SET(1, &cpuset1);
    int s;
    s = pthread_setaffinity_np(pthread_self(), sizeof(cpu_set_t), &cpuset1);
    if (s != 0)
        cout << "Fail to pin to core " << "scheduler" << endl;

    

    for(int i = 0; i < n; i++)
    {
    GPU_task[sched_order[i]-1].ready = true;
        while(GPU_task[sched_order[i]-1].ready == true)
        {
        
        }
    }


    return 0;
}

void * pthread0(void *data)       
{

    struct para* tt = (struct para*)data;

    // pin to a core
    cpu_set_t cpuset1;
    CPU_ZERO(&cpuset1);
    CPU_SET(2+tt->task_num, &cpuset1);
    int s;
    s = pthread_setaffinity_np(pthread_self(), sizeof(cpu_set_t), &cpuset1);
    if (s != 0)
        cout << "Fail to pin to core " << 2+tt->task_num << endl;

  

    hipStream_t stream;
    hipStreamCreate(&stream);


    while(GPU_task[tt->task_num].ready == false)
    {
    }

    
    //cout << "memory length: " << tt->memory_length << endl;

    gettimeofday(&global_tv[tt->task_num],NULL);
    global_memory_start_time[tt->task_num] = global_tv[tt->task_num].tv_sec*1000 + global_tv[tt->task_num].tv_usec/1000 - offset_start_time;

    for(int i = 0; i < tt->memory_length*285; i++)
    {
    hipMemcpyAsync((void*)tt->d_data1, (void*)tt->d_data01, tt->nBytes, hipMemcpyHostToDevice,stream);
    hipMemcpyAsync((void*)tt->d_data2, (void*)tt->d_data02, tt->nBytes, hipMemcpyHostToDevice,stream);
    //hipMemcpyAsync((void*)tt->d_data03, (void*)tt->d_data3, tt->nBytes, hipMemcpyDeviceToHost,stream);
    }
    hipStreamSynchronize(stream);

    GPU_task[tt->task_num].ready = false;



    gettimeofday(&global_tv[tt->task_num],NULL);
    global_memory_finish_time[tt->task_num] = global_tv[tt->task_num].tv_sec*1000 + global_tv[tt->task_num].tv_usec/1000 - offset_start_time;

    GPU_task[tt->task_num].memory_finish = true;


    //---------alg-II--------------------//
    /*
    if((tt->task_num != 0)&&(tt->task_num != 1))
    {
    while((GPU_task[7].memory_finish == false))
    {

    }
    }
    */


    //---------alg-B--------------------//
    ///*
    while((GPU_task[0].memory_finish == false)||(GPU_task[1].memory_finish == false)||(GPU_task[2].memory_finish == false)||(GPU_task[3].memory_finish == false)||(GPU_task[4].memory_finish == false)||(GPU_task[5].memory_finish == false)||(GPU_task[6].memory_finish == false)||(GPU_task[7].memory_finish == false))
    {

    }

    if((tt->task_num == 0)||(tt->task_num == 1))
    {
    while((GPU_task[2].kernel_finish == false)||(GPU_task[3].kernel_finish == false)||(GPU_task[4].kernel_finish == false)||(GPU_task[5].kernel_finish == false)||(GPU_task[6].kernel_finish == false)||(GPU_task[7].kernel_finish == false))
    {

    }
    }
    //*/

    gettimeofday(&global_tv[tt->task_num],NULL);
    global_kernel_start_time[tt->task_num] = global_tv[tt->task_num].tv_sec*1000 + global_tv[tt->task_num].tv_usec/1000 - offset_start_time;

    task <<<tt->gridsize,tt->blocksize,0,stream>>> (tt->d_data1, tt->d_data2, tt->d_data3, tt->N, tt->SM_num_start, tt->SM_num_end, tt->kernel_length);


    hipStreamSynchronize(stream);

    gettimeofday(&global_tv[tt->task_num],NULL);
    global_kernel_finish_time[tt->task_num] = global_tv[tt->task_num].tv_sec*1000 + global_tv[tt->task_num].tv_usec/1000 - offset_start_time;

    GPU_task[tt->task_num].kernel_finish = true;

 
    hipStreamDestroy(stream);

    return 0;
}

