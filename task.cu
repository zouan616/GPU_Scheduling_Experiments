#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include ""
#include <stdint.h>
#include <iostream>
#include <stdlib.h>      
#include <time.h>  
#include <string.h>
#include <limits.h>
#include <math.h>
#include <unistd.h>
#include <fcntl.h>
#include <float.h>
#include <sys/time.h>
#include <fstream>
#include "timespec_functions.h"
#include <algorithm>

using namespace std;

//global function
__global__ void task(float* x, float * y, float* z, int n, int SM_num_start, int SM_num_end){

int SM_num;
SM_num = __mysmid();


if((SM_num_start <= SM_num)&&(SM_num <= SM_num_end))
{    
    // Key technique use the (__mysmid() - SM_num_start) as blockIdx.x
    // global index
    long int index = threadIdx.x + (SM_num - SM_num_start) * blockDim.x;
    // step
    // interleaved execution
    long int off_set = blockDim.x * (SM_num_end - SM_num_start + 1);
    
    if(blockIdx.x < 28)
    {
        for (long int i = index; i < n/2; i += off_set)
        {
            z[i] = x[i] + y[i];
            for(int j = 0; j< 100; j++)
            {
            z[i] = z[i] + x[i] + y[i];
            }
        }
    }
    else
    {
        for (long int i = index + n/2; i < n; i += off_set)
        {
            z[i] = x[i] + y[i];
            for(int j = 0; j< 100; j++)
            {
            z[i] = z[i] + x[i] + y[i];
            }
        }
    }
//printf("SM num: %d ; Block id: %d \n", SM_num, blockIdx.x);
}

}
